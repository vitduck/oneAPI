#include <iostream>
#include <hip/hip_runtime.h> 
#include <cudnn.h>

int main(int argc, char** argv) {
    // device selection  
    hipSetDevice(0); 

    // tensor dimension
    const int N = 1, C = 1, H = 1, W = 7;

    // tensor allocation (USM)
    float *src, *dst; 
    hipMallocManaged(&src, N*C*H*W*sizeof(float));
    hipMallocManaged(&dst, N*C*H*W*sizeof(float));
    
    // tensor initialization
    for (int i=0; i < W; i++) { src[i] = float(i); }

    // filter weigth initialization
    const float alpha = 1.0;
    const float  beta = 0.0;

    // create input tensor descriptor
    cudnnTensorDescriptor_t src_d; 
    cudnnCreateTensorDescriptor(&src_d);
    cudnnSetTensor4dDescriptor(src_d, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, N, C, H, W); 

    // create  output tensor descriptor
    cudnnTensorDescriptor_t dst_d; 
    cudnnCreateTensorDescriptor(&dst_d);
    cudnnSetTensor4dDescriptor(dst_d, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, N, C, H, W); 

    // create activation descriptor
    cudnnActivationDescriptor_t sigmoid_d;
    cudnnCreateActivationDescriptor(&sigmoid_d);
    cudnnSetActivationDescriptor(sigmoid_d, CUDNN_ACTIVATION_SIGMOID, CUDNN_NOT_PROPAGATE_NAN, 0.0f);

    // create cuda handle
    cudnnHandle_t handle;
    cudnnCreate(&handle);

    // sigmoid activation 
    cudnnActivationForward(handle, sigmoid_d, &alpha, src_d, src, &beta, dst_d, dst); 
    
    // free cuDNN
    // data is automatically copied back to host
    cudnnDestroy(handle);
    cudnnDestroyTensorDescriptor(src_d); 
    cudnnDestroyTensorDescriptor(dst_d); 
    cudnnDestroyActivationDescriptor(sigmoid_d); 

    std::cout << "src tensor: "; 
    for (int i=0; i < W; i++) { std::cout << " " << src[i]; } 
    std::cout << std::endl; 
    
    std::cout << "dst tensor: "; 
    for (int i=0; i < W; i++) { std::cout << " " << dst[i]; } 
    std::cout << std::endl; 

    // free memory
    hipFree(src); 
    hipFree(dst); 

    return 0; 
}
