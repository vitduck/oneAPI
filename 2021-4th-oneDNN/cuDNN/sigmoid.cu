#include <iostream>
#include <vector> 
#include <algorithm>
#include <hip/hip_runtime.h> 
#include <cudnn.h>

int main(int argc, char** argv) { 
    // device selection  
    hipSetDevice(0); 

    // tensor dimension
    int    n = 1, c = 1, h = 1, w = 7;
    int size = n * c * h * w; 

    // host allocation 
    std::vector<float> src(size); 
    std::vector<float> dst(size); 

    // tensor initialization
    for (int i=0; i < size; i++) { src[i] = float(i); }

    // tensor allocation on device
    // can't use std::vector directly in cuDNN
    float *ds, *dd; 
    hipMalloc(&ds, src.size()*sizeof(float));
    hipMalloc(&dd, src.size()*sizeof(float));

    // copy src tensor to device memory
    hipMemcpy(ds, src.data(), size*sizeof(float), hipMemcpyHostToDevice);
    
    // filter weigth initialization
    const float alpha = 1.0f; 
    const float beta  = 0.0f; 

    // create input tensor descriptor
    cudnnTensorDescriptor_t src_d; 
    cudnnCreateTensorDescriptor(&src_d);
    cudnnSetTensor4dDescriptor(src_d, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, n, c, h, w); 

    // create output tensor descriptor
    cudnnTensorDescriptor_t dst_d;
    cudnnCreateTensorDescriptor(&dst_d);
    cudnnSetTensor4dDescriptor(dst_d, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, n, c, h, w); 

    // create activation descriptor
    cudnnActivationDescriptor_t sigmoid_d;
    cudnnCreateActivationDescriptor(&sigmoid_d);
    cudnnSetActivationDescriptor(sigmoid_d, CUDNN_ACTIVATION_SIGMOID, CUDNN_NOT_PROPAGATE_NAN, 0.0f);

    // create cuda handle
    cudnnHandle_t handle;
    cudnnCreate(&handle);

    // sigmoid activation 
    cudnnActivationForward( handle, sigmoid_d, &alpha, src_d, ds, &beta, dst_d, dd); 

    // cleanup 
    cudnnDestroy(handle);
    cudnnDestroyTensorDescriptor(src_d);
    cudnnDestroyTensorDescriptor(dst_d);
    cudnnDestroyActivationDescriptor(sigmoid_d);

    // copy dst tensor to host memory
    hipMemcpy(dst.data(), dd, size*sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "src tensor: "; 
    for (int i=0; i < size; i++) { std::cout << " " << src[i]; } 
    std::cout << std::endl; 
    
    std::cout << "dst tensor: "; 
    for (int i=0; i < size; i++) { std::cout << " " << dst[i]; } 
    std::cout << std::endl; 

    // free memory
    hipFree(ds); 
    hipFree(dd); 

    return 0; 
} 
