#include <iostream>
#include <vector> 
#include <algorithm>
#include <hip/hip_runtime.h> 
#include <cudnn.h>

int main(int argc, char** argv) { 
    // device selection  
    hipSetDevice(0); 

    // tensor dimension
    const int N = 1, C = 1, H = 1, W = 7;

    // host allocation 
    std::vector<float> src(N*C*H*W); 
    std::vector<float> dst(N*C*H*W); 

    // tensor initialization
    for (int i=0; i < src.size(); i++) { src[i] = float(i); }

    // tensor allocation on device
    float *ds, *dd; 
    hipMalloc(&ds, src.size()*sizeof(float));
    hipMalloc(&dd, src.size()*sizeof(float));

    // copy src tensor to device memory
    hipMemcpy(ds, src.data(), src.size()*sizeof(float), hipMemcpyHostToDevice);
    
    // filter weigth initialization
    const float alpha = 1.0f; 
    const float beta  = 0.0f; 

    // create input tensor descriptor
    cudnnTensorDescriptor_t src_d; 
    cudnnCreateTensorDescriptor(&src_d);
    cudnnSetTensor4dDescriptor(src_d, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, N, C, H, W); 

    // create output tensor descriptor
    cudnnTensorDescriptor_t dst_d;
    cudnnCreateTensorDescriptor(&dst_d);
    cudnnSetTensor4dDescriptor(dst_d, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, N, C, H, W); 

    // create activation descriptor
    cudnnActivationDescriptor_t sigmoid_d;
    cudnnCreateActivationDescriptor(&sigmoid_d);
    cudnnSetActivationDescriptor(sigmoid_d, CUDNN_ACTIVATION_SIGMOID, CUDNN_NOT_PROPAGATE_NAN, 0.0f);

    // create cuda handle
    cudnnHandle_t handle;
    cudnnCreate(&handle);

    // sigmoid activation 
    cudnnActivationForward( handle, sigmoid_d, &alpha, src_d, ds, &beta, dst_d, dd); 

    // cleanup 
    cudnnDestroy(handle);
    cudnnDestroyTensorDescriptor(src_d);
    cudnnDestroyTensorDescriptor(dst_d);
    cudnnDestroyActivationDescriptor(sigmoid_d);

    // copy dst tensor to host memory
    hipMemcpy(dst.data(), dd, dst.size()*sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "src tensor: "; 
    for (int i=0; i < src.size(); i++) { std::cout << " " << src[i]; } 
    std::cout << std::endl; 
    
    std::cout << "dst tensor: "; 
    for (int i=0; i < dst.size(); i++) { std::cout << " " << dst[i]; } 
    std::cout << std::endl; 

    // free memory
    hipFree(ds); 
    hipFree(dd); 

    return 0; 
} 
