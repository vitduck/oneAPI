#include <iostream>
#include <hipDNN.h>
#include <opencv2/core.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/imgcodecs.hpp>

int main(int argc, char** argv) { 
    // convert src image to matrix format 
    cv::Mat src = cv::imread("./Lenna.png", cv::IMREAD_COLOR);

    // convert src matrix to FP32 with 3 channels
    src.convertTo(src, CV_32FC3); 

    // normalize color range to (0,1)
    cv::normalize(src, src, 0, 1, cv::NORM_MINMAX); 

    // image dimension 
    int channel = src.channels(); 
    int height  = src.rows; 
    int width   = src.cols; 
    int size    = channel * height * width * sizeof(float); 

    // dst vector allocation 
    float *dst = (float*) malloc(size); 
    cv::Mat edge(height, width, CV_32FC3, dst); 

    // filter weigth 
    const float alpha = 1.0f;
    const float beta  = 0.0f;

    // edge detection kernel 
    const float laplacian[3][3] = { {1, 1, 1}, {1, -8, 1}, {1, 1, 1} };

    // 4d conv kernel 
    float filter[3][3][3][3];

    // copy laplacian to 4d tensor
    for (int i = 0; i < 3; i++) 
        for (int j = 0; j < 3; j++) 
            for (int k = 0; k < 3; k++) 
                for (int m = 0; m < 3; m++) 
                    filter[i][j][k][m] = laplacian[k][m];

    // device selection  
    hipSetDevice(0); 

    // create cuda handle
    hipdnnHandle_t handle;
    hipdnnCreate(&handle);

    // create input tensor descriptor
    hipdnnTensorDescriptor_t src_d; 
    hipdnnCreateTensorDescriptor(&src_d);
    hipdnnSetTensor4dDescriptor(src_d, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, 1, channel, height, width); 

    // create filter descriptor 
    hipdnnFilterDescriptor_t filter_d; 
    hipdnnCreateFilterDescriptor(&filter_d); 
    hipdnnSetFilter4dDescriptor(filter_d, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 3, 3, 3, 3); 

    // create convolution descriptor 
    hipdnnConvolutionDescriptor_t conv_d; 
    hipdnnCreateConvolutionDescriptor(&conv_d); 
    hipdnnSetConvolution2dDescriptor(conv_d, 1, 1, 1, 1, 1, 1, HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT); 

    // create output tensor descriptor
    hipdnnTensorDescriptor_t dst_d; 
    hipdnnCreateTensorDescriptor(&dst_d);
    hipdnnSetTensor4dDescriptor(dst_d, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, 1, channel, height, width); 

    // creat convulution algorithm 
    hipdnnConvolutionFwdAlgo_t conv_algo; 
    hipdnnGetConvolutionForwardAlgorithm(handle, src_d, filter_d, conv_d, dst_d, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &conv_algo); 

    // get workspace size
    size_t ws_size; 
    hipdnnGetConvolutionForwardWorkspaceSize(handle, src_d, filter_d, conv_d, dst_d, conv_algo, &ws_size); 

    // memory allocation for input/output/filter/workspace
    void *ds, *dd, *df, *dw; 
    hipMalloc(&ds, size);
    hipMalloc(&dd, size);
    hipMalloc(&df, size);
    hipMalloc(&dw, ws_size);

    // copy data to device 
    hipMemcpy(ds, src.ptr<float>(0), size, hipMemcpyHostToDevice);
    hipMemcpy(df, filter, sizeof(filter)*sizeof(float), hipMemcpyHostToDevice);

    // convolution 
    hipdnnConvolutionForward(handle, &alpha, src_d, ds, filter_d, df, conv_d, conv_algo, dw, ws_size, &beta, dst_d, dd); 

    // copy data back to host 
    hipMemcpy(dst, dd, size, hipMemcpyDeviceToHost);

    // avoid negative pixels ?! 
    cv::threshold(edge, edge, 0, 0, cv::THRESH_TOZERO);
   
    // renormalize to RBG range 
    cv::normalize(edge, edge, 0.0, 255.0, cv::NORM_MINMAX);

    // convert back to 8bit (RGB) format 
    edge.convertTo(edge, CV_8UC3);

    // write to png file 
    cv::imwrite("detection.png", edge);

    // clean up dnn
    hipdnnDestroy(handle);
    hipdnnDestroyTensorDescriptor(src_d);
    hipdnnDestroyTensorDescriptor(dst_d);
    hipdnnDestroyFilterDescriptor(filter_d);
    hipdnnDestroyConvolutionDescriptor(conv_d); 

    // free memory
    hipFree(ds); 
    hipFree(dd); 
    hipFree(df); 
    hipFree(dw); 
}
