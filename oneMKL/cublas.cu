#include <iostream>
#include <cstdlib>
#include <cstring>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define SEED 666
#define SIZE 16834
#define LOOP 100

#include "util.hpp"

int main(int argc, char *argv[]) {
    hipblasStatus_t status; 

    // scalar multiplier
    float alpha = 1.0, beta = 1.0;

    // matrix size (squared) 
    int m = SIZE, n = SIZE, k = SIZE; 

    // leading dimension 
    int ldA = k , ldB = n, ldC = n;  

    // host data
    float* A = (float *) aligned_alloc(32, (m * k) * sizeof(float));
    float* B = (float *) aligned_alloc(32, (k * n) * sizeof(float));
    float* C = (float *) aligned_alloc(32, (m * n) * sizeof(float));

    // create random square matrix 
    srand(SEED); 
    random_matrix<float>(A, m, k); 
    random_matrix<float>(B, k, n); 
    zero_matrix<float>(C, m, n); 

    // device data
    float *dA, *dB, *dC; 
    hipMalloc((void**) &dA, (m * k) * sizeof(float));
    hipMalloc((void**) &dB, (k * n) * sizeof(float));
    hipMalloc((void**) &dC, (m * n) * sizeof(float));

    // copy matrix to gpu
    hipblasSetMatrix(m, k, sizeof(float), A, ldA, dA, ldA); 
    hipblasSetMatrix(k, n, sizeof(float), B, ldB, dB, ldB); 
    hipblasSetMatrix(m, n, sizeof(float), C, ldC, dC, ldC); 
    
    // cublas context
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // warmup 
    status = hipblasSgemm(
        handle, 
        HIPBLAS_OP_T, HIPBLAS_OP_T,
        m, n, k, 
        &alpha, dA, ldA, 
        dB, ldB, 
        &beta, dC, ldC
    ); 
    // copy matrix to host
    hipblasGetMatrix(m, n, sizeof(float), dC, ldC, C, ldC); 

    // real measurement 
    std::cout << "SGEMM using native cuBLAS" << std::endl; 
    std::cout << "Matrix size: " << SIZE << std::endl; 

    auto start = std::chrono::system_clock::now(); 
    for (int i=0; i < LOOP; i++) { 
        // warmup 
        status = hipblasSgemm(
            handle, 
            HIPBLAS_OP_T, HIPBLAS_OP_T,
            m, n, k, 
            &alpha, dA, ldA, 
            dB, ldB, 
            &beta, dC, ldC
        ); 
        // copy matrix to host
        hipblasGetMatrix(m, n, sizeof(float), dC, ldC, C, ldC); 
    } 
    auto end = std::chrono::system_clock::now(); 

    hipblasDestroy(handle); 

    // walltime 
    std::chrono::duration<float> walltime = end-start;

    // gflops 
    float average = walltime.count()/LOOP;  
    float gflops  = 2.0*m*n*k*1E-9/average; 

    std::cout << "Average running time: " << average << std::endl; 
    std::cout << "Performance GFLOPS: "   << gflops  << std::endl; 

    free(A); 
    free(B); 
    free(C); 
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    
    return 0;
}
