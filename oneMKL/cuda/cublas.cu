#include <iostream>
#include <cstdlib>
#include <cstring>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "util.hpp"

#define SEED 666

int SIZE = 4096; 
int LOOP = 100; 

int main(int argc, char *argv[]) {
    // getopt
    parseArguments(argc, argv); 

    // scalar multiplier
    float alpha = 1.0, beta = 1.0;

    // matrix size (squared) 
    int m = SIZE, n = SIZE, k = SIZE; 

    // leading dimension 
    int ldA = k , ldB = n, ldC = n;  

    // host data
    float* A = (float *) aligned_alloc(32, (m * k) * sizeof(float));
    float* B = (float *) aligned_alloc(32, (k * n) * sizeof(float));
    float* C = (float *) aligned_alloc(32, (m * n) * sizeof(float));

    // create random square matrix 
    srand(SEED); 
    random_matrix<float>(A, m, k); 
    random_matrix<float>(B, k, n); 
    zero_matrix<float>(C, m, n); 

    // device data
    float *dA, *dB, *dC; 
    hipMalloc((void**) &dA, (m * k) * sizeof(float));
    hipMalloc((void**) &dB, (k * n) * sizeof(float));
    hipMalloc((void**) &dC, (m * n) * sizeof(float));

    // copy matrix to gpu
    hipblasSetMatrix(m, k, sizeof(float), A, ldA, dA, ldA); 
    hipblasSetMatrix(k, n, sizeof(float), B, ldB, dB, ldB); 
    hipblasSetMatrix(m, n, sizeof(float), C, ldC, dC, ldC); 
    
    // cublas context
    hipblasStatus_t status; 
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // cuda events 
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // warmup 
    status = hipblasSgemm(
        handle, 
        HIPBLAS_OP_T, HIPBLAS_OP_T,
        m, n, k, 
        &alpha, dA, ldA, 
        dB, ldB, 
        &beta, dC, ldC
    ); 
    
    // real measurement 
    std::cout << "SGEMM using native cuBLAS" << std::endl; 
    std::cout << "Matrix size: " << SIZE << std::endl; 
    std::cout << "Loop count: "  << LOOP << std::endl; 

    hipEventRecord(start); 
    for (int i=0; i < LOOP; i++) { 
        status = hipblasSgemm(
            handle, 
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            m, n, k, 
            &alpha, dA, ldA, 
            dB, ldB, 
            &beta, dC, ldC
        ); 
    } 
    hipEventRecord(stop);
        
    // copy matrix back to host
    hipblasGetMatrix(m, n, sizeof(float), dC, ldC, C, ldC); 

    hipblasDestroy(handle); 
    
    // walltime 
    float walltime = 0; 
    hipEventElapsedTime(&walltime, start, stop);

    // gflops 
    float average = 0.001*walltime/LOOP;  
    float gflops  = 2.0*m*n*k*1E-9/average; 

    std::cout << "Average running time: " << average << std::endl; 
    std::cout << "Performance GFLOPS: "   << gflops  << std::endl; 

    free(A); 
    free(B); 
    free(C); 
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    
    return 0;
}
